#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


#define BLOCKSIZE 256

__global__ void MatrixAddI(int *matrix1, int *matrix2, int *matrix3, int m, int n)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x < m*n)
  {
    matrix3[x] = matrix1[x] + matrix2[x];
  }
}

__global__ void MatrixAddF(float *matrix1, float *matrix2, float *matrix3, int m, int n)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;

  if (x < m*n)
  {
    matrix3[x] = matrix1[x] + matrix2[x];
  }
}

void read_imatrix(char *filename, int *m, int *n, int **values)
{
  FILE* name;
  int i, j, k;
  int t1, t2, t3;
  name = fopen(filename, "r+");
  if(name != NULL)
  {
    k = 0;
    fscanf(name, "%d %d\n", &t1, &t2);
    *m = t1;
    *n = t2;
    *values = (int *)calloc(t1*t2, sizeof(int));
    for(i = 1; i <= t1; i++)
    {
      for(j = 1; j <= t2; j++)
      {
        if(j < t2)
        {
          fscanf(name, "%d,", &t3);
          *(*values+k) = t3;
          k++;
        }
        else
        {
          fscanf(name, "%d\n", &t3);
          *(*values+k) = t3;
          k++;
        }
      }
    }
    fclose(name);
  }
  else
  {
    printf("File read failed\n");
    exit(1);
  }
}

void read_fmatrix(char *filename, int *m, int *n, float **values)
{
  FILE* name;
  int i, j, k;
  int t1, t2;
  float t3;
  name = fopen(filename, "r+");
  if(name != NULL)
  {
    k = 0;
    fscanf(name, "%d %d\n", &t1, &t2);
    *m = t1;
    *n = t2;
    *values = (float *)calloc(t1*t2, sizeof(float));
    for(i = 1; i <= t1; i++)
    {
      for(j = 1; j <= t2; j++)
      {
        if(j < t2)
        {
          fscanf(name, "%f,", &t3);
          *(*values+k) = t3;
          k++;
        }
        else
        {
          fscanf(name, "%f\n", &t3);
          *(*values+k) = t3;
          k++;
        }
      }
    }
    fclose(name);
  }
  else
  {
    printf("File read failed\n");
    exit(1);
  }
}

void write_imatrix(char *filename, int *m, int *n, int **values)
{
  FILE* name;
  int i, j, k;
  int t1, t2, t3;
  name = fopen(filename, "w+");
  if(name != NULL)
  {
    k = 0;
    t1 = *m;
    t2 = *n;
    fprintf(name, "%d %d\n", t1, t2);
    for(i = 1; i <= t1; i++)
    {
      for(j = 1; j <= t2; j++)
      {
        if(j < t2)
        {
          t3 = *(*values+k);
          fprintf(name, "%d,", t3);
          k++;
        }
        else
        {
          t3 = *(*values+k);
          fprintf(name, "%d\n", t3);
          k++;
        }
      }
    }
    fclose(name);
  }
  else
  {
    printf("File write failed\n");
    exit(1);
  }
}

void write_fmatrix(char *filename, int *m, int *n, float **values)
{
  FILE* name;
  int i, j, k;
  int t1, t2;
  float t3;
  name = fopen(filename, "w+");
  if(name != NULL)
  {
    k = 0;
    t1 = *m;
    t2 = *n;
    fprintf(name, "%d %d\n", t1, t2);
    for(i = 1; i <= t1; i++)
    {
      for(j = 1; j <= t2; j++)
      {
        if(j < t2)
        {
          t3 = *(*values+k);
          fprintf(name, "%f,", t3);
          k++;
        }
        else
        {
          t3 = *(*values+k);
          fprintf(name, "%f\n", t3);
          k++;
        }
      }
    }
    fclose(name);
  }
  else
  {
    printf("File write failed\n");
    exit(1);
  }
}

void matrix_check(int m1, int n1, int m2, int n2)
{
  if ((m1-m2)+(n1-n2) != 0)
  {
    printf("Matrix dimensions must be PxQ and PxQ respectively\n");
    exit(1);
  }
}

int main(int argc, char *argv[])
{
  int m1, n1, m2, n2;

  if (argc != 5)
  {
    printf("Usage: ./matrix-addition matrix1.mat matrix2.mat matrix3.mat float/int \n");
    exit(1);
  }

  if (strcmp(argv[4], "float") == 0)
  {
    float *hostmatrix1, *hostmatrix2, *hostmatrix3;
    float *devicematrix1, *devicematrix2, *devicematrix3;
    int GRIDSIZE;
    read_fmatrix(argv[1], &m1, &n1, &hostmatrix1);
    read_fmatrix(argv[2], &m2, &n2, &hostmatrix2);
    matrix_check(m1, n1, m2, n2);
    size_t matrix_size = m1*n1*sizeof(float);
    hostmatrix3 = (float *)calloc(matrix_size, sizeof(float));
    hipMalloc(&devicematrix1, matrix_size);
    hipMalloc(&devicematrix2, matrix_size);
    hipMalloc(&devicematrix3, matrix_size);
    hipMemcpy(devicematrix1, hostmatrix1, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(devicematrix2, hostmatrix2, matrix_size, hipMemcpyHostToDevice);
    GRIDSIZE = (int)ceil((float)(m1*n1)/BLOCKSIZE);
    dim3 dimGrid(GRIDSIZE, 1, 1);
    dim3 dimBlock(BLOCKSIZE, 1, 1);
    MatrixAddF <<< dimGrid, dimBlock >>> (devicematrix1, devicematrix2, devicematrix3, m1, n1);
    hipMemcpy(hostmatrix3, devicematrix3, matrix_size, hipMemcpyDeviceToHost);
    write_fmatrix(argv[3], &m1, &n1, &hostmatrix3);
    hipFree(devicematrix1);
    hipFree(devicematrix2);
    hipFree(devicematrix3);
    free(hostmatrix1);
    free(hostmatrix2);
    free(hostmatrix3);
  }

  if (strcmp(argv[4], "int") == 0)
  {
    int *hostmatrix1, *hostmatrix2, *hostmatrix3;
    int *devicematrix1, *devicematrix2, *devicematrix3;
    int GRIDSIZE;
    read_imatrix(argv[1], &m1, &n1, &hostmatrix1);
    read_imatrix(argv[2], &m2, &n2, &hostmatrix2);
    matrix_check(m1, n1, m2, n2);
    size_t matrix_size = m1*n1*sizeof(int);
    hostmatrix3 = (int *)calloc(m1*n1, sizeof(int));
    hipMalloc(&devicematrix1, matrix_size);
    hipMalloc(&devicematrix2, matrix_size);
    hipMalloc(&devicematrix3, matrix_size);
    hipMemcpy(devicematrix1, hostmatrix1, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(devicematrix2, hostmatrix2, matrix_size, hipMemcpyHostToDevice);
    GRIDSIZE = (int)ceil((float)(m1*n1)/BLOCKSIZE);
    dim3 dimGrid(GRIDSIZE, 1, 1);
    dim3 dimBlock(BLOCKSIZE, 1, 1);
    MatrixAddI <<< dimGrid, dimBlock >>> (devicematrix1, devicematrix2, devicematrix3, m1, n1);
    hipMemcpy(hostmatrix3, devicematrix3, matrix_size, hipMemcpyDeviceToHost);
    write_imatrix(argv[3], &m1, &n1, &hostmatrix3);
    hipFree(devicematrix1);
    hipFree(devicematrix2);
    hipFree(devicematrix3);
    free(hostmatrix1);
    free(hostmatrix2);
    free(hostmatrix3);
  }

  return 0;
}
